#include "hip/hip_runtime.h"
#include <cstdio>
#include "../common.h"
#include "pim_real.h"

__global__ void bn_fwd(row_t *mem_rows, int num_rows, row_t *mean,
        row_t *var, row_t *weight, row_t *bias, row_t *eps, row_t *temp) {
    // This is the maximum number of threads that will issue requests for a
    // group of banks that map to a unique PIM unit
    int threads_per_pim_grp = NUM_THREADS_PER_WARP / \
                              (NUM_BANKS / NUM_PIM_UNITS);

    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id_in_grp = thread_id % threads_per_pim_grp;

    int chip = thread_id / NUM_THREADS_PER_WARP;
    int bank = ((thread_id % NUM_THREADS_PER_WARP) / threads_per_pim_grp) * \
               NUM_PIM_UNITS;

    float dummy = 100;
    uint16_t store = 100;

    // Compute invstd (temp[0])
    for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {

        // reg = var
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(var->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // reg = reg + eps
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(eps->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // reg = sqrt(reg)
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(var->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // reg = 1 / reg
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(var->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // temp[0] = reg
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.u16 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(temp[0].val[mem_index])), "h"(store)
                    : /* no clobbers */);
        }

        __threadfence();
    }

    // Compute gamma (temp[1])
    for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {

        // reg = abs(weight)
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(weight->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // reg = reg + eps
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(eps->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // temp[1] = reg
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.u16 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(temp[1].val[mem_index])), "h"(store)
                    : /* no clobbers */);
        }

        __threadfence();
    }

    for (int row_X = 0; row_X < num_rows; row_X++) {
        int row_z = row_X + num_rows;
        for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {

            // reg = a[i]
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_X].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg - mean
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mean->val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * invstd
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[0].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * gamma
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[1].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg + beta
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(bias->val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // b[i] = reg
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.u16 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_z].val[mem_index])), "h"(store)
                        : /* no clobbers */);
            }

            __threadfence();
        }
    }
}

// mem_rows[0] = z
// mem_rows[1] = dz
// mem_rows[2] = w
// mem_rows[3] = dw
// mem_rows[4] = dx
//
// temp[0] = invstd
// temp[1] = gamma
// temp[2] = mul
// temp[3] = (z - beta) / gamma
// temp[4] and temp[5] change values over time
__global__ void bn_bwd(row_t *mem_rows, int num_rows, row_t *mean,
        row_t *var, row_t *weight, row_t *bias, row_t *eps, row_t *edz,
        row_t *eydz, row_t *temp) {
    // This is the maximum number of threads that will issue requests for a
    // group of banks that map to a unique PIM unit
    int threads_per_pim_grp = NUM_THREADS_PER_WARP / \
                              (NUM_BANKS / NUM_PIM_UNITS);

    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id_in_grp = thread_id % threads_per_pim_grp;

    int chip = thread_id / NUM_THREADS_PER_WARP;
    int bank = ((thread_id % NUM_THREADS_PER_WARP) / threads_per_pim_grp) * \
               NUM_PIM_UNITS;

    float dummy = 100;
    uint16_t store = 100;

    // Compute invstd (temp[0])
    for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {

        // reg = var
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(var->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // reg = reg + eps
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(eps->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // reg = sqrt(reg)
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(var->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // reg = 1 / reg
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(var->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // temp[0] = reg
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.u16 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(temp[0].val[mem_index])), "h"(store)
                    : /* no clobbers */);
        }

        __threadfence();
    }

    // Compute gamma (temp[1])
    for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {

        // reg = abs(weight)
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(weight->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // reg = reg + eps
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(eps->val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // temp[1] = reg
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.u16 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(temp[1].val[mem_index])), "h"(store)
                    : /* no clobbers */);
        }

        __threadfence();
    }

    // Compute mul (temp[2])
    for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {

        // reg = gamma
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(temp[1].val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // reg = reg * invstd
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.f32 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(temp[0].val[mem_index])), "f"(dummy)
                    : /* no clobbers */);
        }

        __threadfence();

        // mul = reg
        for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                i += threads_per_pim_grp) {
            uint64_t mem_index = INDEX(chip, bank, col + i);

            asm volatile ("st.cs.global.u16 [%0], %1;"
                    : /* no outputs */
                    : "l"(&(temp[2].val[mem_index])), "h"(store)
                    : /* no clobbers */);
        }

        __threadfence();
    }

    // Main loop
    for (int row_z = 0; row_z < num_rows; row_z++) {
        int row_dz = row_z  + num_rows;
        int row_w  = row_dz + num_rows;
        int row_dw = row_w  + num_rows;
        int row_dx = row_dw + num_rows;

        for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {

            // reg = z[i]
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_z].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg - bias
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(bias->val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg / gamma
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[1].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // temp[3] = reg
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.u16 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[3].val[mem_index])), "h"(store)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = w[i]
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_w].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * edz
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(edz->val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // temp[4] = reg
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.u16 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[4].val[mem_index])), "h"(store)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = w[i]
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_w].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * temp[3]
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[3].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * eydz
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(eydz->val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // temp[5] = reg
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.u16 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[5].val[mem_index])), "h"(store)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = dz[i]
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_dz].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg - temp[4]
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[4].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg - temp[5]
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[5].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * temp[2]
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[2].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // dx = reg
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.u16 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_dx].val[mem_index])), "h"(store)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = mean
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mean->val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * invstd (temp[0])
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[0].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg + y (temp[3])
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[3].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * edz
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(edz->val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // temp[4] = reg
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.u16 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[4].val[mem_index])), "h"(store)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = y (temp[3])
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[3].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * y (temp[3])
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[3].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * 0.5
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[3].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * eydz
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(eydz->val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg + temp[4]
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[4].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg + mulW (gamma, temp[1])
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(temp[1].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // dw = reg
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.u16 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_dw].val[mem_index])), "h"(store)
                        : /* no clobbers */);
            }

            __threadfence();
        }

        // dw = -dw
        for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {

            // reg = dw
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_dw].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = reg * -1
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_dw].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // mul = reg
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.u16 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row_dw].val[mem_index])), "h"(store)
                        : /* no clobbers */);
            }

            __threadfence();
        }
    }
}

// The number of datapoints is a multiple of row size.
// number of rows = number of points * number of features
__global__ void kmeans(row_t *mem_rows, int num_rows,
        int *cluster_assignment, float *centroids, int *cluster_size,
        int num_datapoints, int num_features, int num_iters, int num_threads,
        row_t *temp)
{
    // This is the maximum number of threads that will issue requests for a
    // group of banks that map to a unique PIM unit
    int threads_per_pim_grp = NUM_THREADS_PER_WARP / \
                              (NUM_BANKS / NUM_PIM_UNITS);

    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id_in_grp = thread_id % threads_per_pim_grp;

    int chip = thread_id / NUM_THREADS_PER_WARP;
    int bank = ((thread_id % NUM_THREADS_PER_WARP) / threads_per_pim_grp) * \
               NUM_PIM_UNITS;

    int num_datapoint_rows = num_rows / num_features;
    int num_datapoints_per_row = (ROW_SIZE * NUM_BANKS * NUM_CHIPS) / 4;

    float dummy = 100;
    uint16_t store = 100;

    for (int iter = 0; iter < num_iters; iter++) {

        for (int c = 0; c < KMEANS_NUM_CLUSTERS; c++) {
            for (int row = 0; row < num_rows; row += num_features) {
                int c_row = (c * num_datapoint_rows) + row;

                for (int f = 0; f < num_features; f++) {
                    for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {

                        // reg = datapoint[f] - centroid[f] (scalar memory)
                        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                                i += threads_per_pim_grp) {
                            uint64_t mem_index = INDEX(chip, bank, col + i);

                            asm volatile ("st.cs.global.f32 [%0], %1;"
                                    : /* no outputs */
                                    : "l"(&(mem_rows[row + f].val[mem_index])),
                                      "f"(dummy)
                                    : /* no clobbers */);
                        }

                        __threadfence();

                        // reg = reg * reg
                        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                                i += threads_per_pim_grp) {
                            uint64_t mem_index = INDEX(chip, bank, col + i);

                            asm volatile ("st.cs.global.f32 [%0], %1;"
                                    : /* no outputs */
                                    : "l"(&(mem_rows[row + f].val[mem_index])),
                                      "f"(dummy)
                                    : /* no clobbers */);
                        }

                        __threadfence();

                        // reg = reg + temp[c]
                        for (int i = thread_id_in_grp; i < PIM_RF_SIZE ;
                                i += threads_per_pim_grp) {
                            uint64_t mem_index = INDEX(chip, bank, col + i);

                            asm volatile ("st.cs.global.f32 [%0], %1;"
                                    : /* no outputs */
                                    : "l"(&(temp[c_row].val[mem_index])),
                                      "f"(dummy)
                                    : /* no clobbers */);
                        }

                        __threadfence();

                        // temp[c] = reg
                        for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                                i += threads_per_pim_grp) {
                            uint64_t mem_index = INDEX(chip, bank, col + i);

                            asm volatile ("st.cs.global.u16 [%0], %1;"
                                    : /* no outputs */
                                    : "l"(&(temp[c_row].val[mem_index])),
                                      "h"(store)
                                    : /* no clobbers */);
                        }

                        __threadfence();
                    }
                }

                // temp[c] = sqrt(temp[c])
                for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {
                    // reg = sqrt(temp[c])
                    for (int i = thread_id_in_grp; i < (PIM_RF_SIZE/2) ;
                            i += threads_per_pim_grp) {
                        uint64_t mem_index = INDEX(chip, bank, col + i);

                        asm volatile ("st.cs.global.f32 [%0], %1;"
                                : /* no outputs */
                                : "l"(&(temp[c_row].val[mem_index])),
                                  "f"(dummy)
                                : /* no clobbers */);
                    }

                    __threadfence();

                    // temp[c] = reg
                    for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                            i += threads_per_pim_grp) {
                        uint64_t mem_index = INDEX(chip, bank, col + i);

                        asm volatile ("st.cs.global.u16 [%0], %1;"
                                : /* no outputs */
                                : "l"(&(temp[c_row].val[mem_index])),
                                  "h"(store)
                                : /* no clobbers */);
                    }

                    __threadfence();
                }
            }
        }

        // Find closest centroid to each datapoint
        for (int p = thread_id; p < num_datapoints; p += num_threads) {
            float min_dist = INFINITY;
            int closest_centroid = 0;

            for(int c = 0; c < KMEANS_NUM_CLUSTERS; c++) {
                int row = (c * num_datapoint_rows) + \
                          (p / num_datapoints_per_row);
                int index = (p % num_datapoints_per_row) * 4;
                float dist = temp[row].val[index];

                if (dist < min_dist) {
                    min_dist = dist;
                    closest_centroid = c;
                }
            }

            //assign closest cluster id for this datapoint/thread
            cluster_assignment[p] = closest_centroid;
        }

        //// Reset cluster sizes and centroids
        //if (thread_id < KMEANS_NUM_CLUSTERS) {
        //    cluster_size[thread_id] = 0;
        //    for (int f = 0; f < num_features; f++) {
        //        centroids[(thread_id * num_features) + f] = 0;
        //    }
        //}

        //__syncthreads();

        //// Compute cluster size
        //float cluster_size_local[KMEANS_NUM_CLUSTERS] = {0};

        //for (int p = threadIdx.x; p < num_datapoints; p += num_threads) {
        //    cluster_size_local[cluster_assignment[p]]++;
        //}

        //for (int c = 0; c < KMEANS_NUM_CLUSTERS; c++) {
        //    atomicAdd(&cluster_size[c], cluster_size_local[c]);
        //}

        //// Compute cluster datapoint sums
        //for (int f = 0; f < num_features; f++) {
        //    float cluster_datapoint_sums[KMEANS_NUM_CLUSTERS] = {0};

        //    for (int p = threadIdx.x; p < num_datapoints; p += num_threads) {
        //        int row = ((p / num_datapoints_per_row) * num_features) + f;
        //        int index = (p % num_datapoints_per_row) * 4;
        //        cluster_datapoint_sums[cluster_assignment[p]] += \
        //                mem_rows[row].val[index];
        //    }

        //    for (int c = 0; c < KMEANS_NUM_CLUSTERS; c++) {
        //        atomicAdd(&centroids[(c * num_features) + f],
        //                cluster_datapoint_sums[c]);
        //    }
        //}

        //__syncthreads();
        //
        //// Recompute centroids
        //if (thread_id < KMEANS_NUM_CLUSTERS) {
        //    for (int f = 0; f < num_features; f++) {
        //        centroids[(thread_id * num_features) + f] /= \
        //                cluster_size[thread_id];
        //    }
        //}

        //__syncthreads();
    }
}

__global__ void histogram(uint32_t *input, int num_elements, row_t *local_bins,
        int num_rows, uint32_t *bins, int num_threads)
{
    // This is the maximum number of threads that will issue requests for a
    // group of banks that map to a unique PIM unit
    int threads_per_pim_grp = NUM_THREADS_PER_WARP / \
                              (NUM_BANKS / NUM_PIM_UNITS);

    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id_in_grp = thread_id % threads_per_pim_grp;

    int chip = thread_id / NUM_THREADS_PER_WARP;
    int bank = ((thread_id % NUM_THREADS_PER_WARP) / threads_per_pim_grp) * \
               NUM_PIM_UNITS;

    float dummy = 100;
    uint16_t store = 100;

    // Initialize local memory to store thread totals
    uint32_t thread_bins[HISTOGRAM_NUM_BINS];
    for (int i = 0; i < HISTOGRAM_NUM_BINS; i++) {
        thread_bins[i] = 0;
    }

    // Compute thread totals
    for (int i = thread_id; i < num_elements; i += num_threads) {
        thread_bins[input[i]]++;
    }

    __syncthreads();

    // Write block-wise totals to global memory
    int row_factor = ROW_SIZE * NUM_BANKS * NUM_CHIPS;
    int bin_row = (thread_id * HISTOGRAM_NUM_BINS * 4) / row_factor;
    int base_index = (thread_id * HISTOGRAM_NUM_BINS * 4) % row_factor;

    for (int i = 0; i < HISTOGRAM_NUM_BINS; i++) {
        local_bins[bin_row].val[base_index + (i * 4)] = thread_bins[i];
    }

    __syncthreads();

    // Sum values from all blocks
    int result_row = num_rows - 1;
    for (int row = 0; row < result_row; row++) {
        for (int bin = 0; bin < ((NUM_COLS * COL_SIZE) / 4);
                bin += HISTOGRAM_NUM_BINS) {
            for (int col = 0; col < HISTOGRAM_NUM_BINS; col += PIM_RF_SIZE) {

                // reg = local_bin
                for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                        i += threads_per_pim_grp) {
                    uint64_t mem_index = INDEX(chip, bank, bin + col + i);

                    asm volatile ("st.cs.global.f32 [%0], %1;"
                            : /* no outputs */
                            : "l"(&(local_bins[row].val[mem_index])),
                              "f"(dummy)
                            : /* no clobbers */);
                }

                __threadfence();

                // reg = reg + bin_total
                for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                        i += threads_per_pim_grp) {
                    uint64_t mem_index = INDEX(chip, bank, col + i);

                    asm volatile ("st.cs.global.f32 [%0], %1;"
                            : /* no outputs */
                            : "l"(&(local_bins[result_row].val[mem_index])),
                              "f"(dummy)
                            : /* no clobbers */);
                }

                __threadfence();

                // bin_total = reg
                for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                        i += threads_per_pim_grp) {
                    uint64_t mem_index = INDEX(chip, bank, col + i);

                    asm volatile ("st.cs.global.u16 [%0], %1;"
                            : /* no outputs */
                            : "l"(&(local_bins[result_row].val[mem_index])),
                              "h"(store)
                            : /* no clobbers */);
                }

                __threadfence();
            }
        }
    }

    if (thread_id < HISTOGRAM_NUM_BINS) {
        uint32_t total = 0;

        for (int c = 0; c < NUM_CHIPS; c++) {
            for (int b = 0; b < NUM_BANKS; b++) {
                for (int bin = 0; bin < ((NUM_COLS * COL_SIZE) / 4);
                        bin += HISTOGRAM_NUM_BINS) {
                    uint64_t mem_index = INDEX(c, b, bin + thread_id);
                    total += local_bins[result_row].val[mem_index];
                }
            }
        }

        bins[thread_id] = total;
    }
}

/*
 * Each row is the first element of all vectors within the batch.
 * This means that:
 * 1) num_rows = num elements in each vector
 * 2) row size = batch size
 */
__global__ void fully_connected(row_t *input, row_t *weights, row_t *output,
        int num_rows, int num_inputs, int num_outputs)
{
    // This is the maximum number of threads that will issue requests for a
    // group of banks that map to a unique PIM unit
    int threads_per_pim_grp = NUM_THREADS_PER_WARP / \
                              (NUM_BANKS / NUM_PIM_UNITS);

    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id_in_grp = thread_id % threads_per_pim_grp;

    int chip = thread_id / NUM_THREADS_PER_WARP;
    int bank = ((thread_id % NUM_THREADS_PER_WARP) / threads_per_pim_grp) * \
               NUM_PIM_UNITS;

    float dummy = 100;
    uint16_t store = 100;

    for (int out = 0; out < num_outputs; out++) {
        for (int col = 0; col < NUM_COLS; col += (PIM_RF_SIZE / 2)) {
            // reg[0] = weights
            for (int i = thread_id_in_grp; i < (PIM_RF_SIZE / 2);
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(weights[out].val[mem_index])),
                          "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            for (int row = 0; row < num_rows; row++) {
                // reg[1] = reg[1] + (a[i] * reg[0])
                for (int i = thread_id_in_grp; i < (PIM_RF_SIZE / 2);
                        i += threads_per_pim_grp) {
                    uint64_t mem_index = INDEX(chip, bank, col + i);

                    asm volatile ("st.cs.global.f32 [%0], %1;"
                            : /* no outputs */
                            : "l"(&(input[row].val[mem_index])),
                              "f"(dummy)
                            : /* no clobbers */);
                }

                __threadfence();
            }

            // reg[1] = reg[1] + bias (scalar)
            for (int i = thread_id_in_grp; i < (PIM_RF_SIZE / 2);
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(output[out].val[mem_index])),
                          "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // output[i] = reg[1]
            for (int i = thread_id_in_grp; i < (PIM_RF_SIZE / 2);
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.u16 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(output[out].val[mem_index])),
                          "h"(store)
                        : /* no clobbers */);
            }

            __threadfence();
        }
    }
}

__global__ void grim(row_t *mem_rows, int num_rows)
{
    // This is the maximum number of threads that will issue requests for a
    // group of banks that map to a unique PIM unit
    int threads_per_pim_grp = NUM_THREADS_PER_WARP / \
                              (NUM_BANKS / NUM_PIM_UNITS);

    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_id_in_grp = thread_id % threads_per_pim_grp;

    int chip = thread_id / NUM_THREADS_PER_WARP;
    int bank = ((thread_id % NUM_THREADS_PER_WARP) / threads_per_pim_grp) * \
               NUM_PIM_UNITS;

    float dummy = 100;
    uint16_t store = 100;

    int mask_row = num_rows - 1;
    for (int row = 0; row < mask_row; row++) {
        for (int col = 0; col < NUM_COLS; col += PIM_RF_SIZE) {

            // reg = GRIM(input[i])
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[row].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            __threadfence();

            // reg = result[i] OR mask
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.f32 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[mask_row].val[mem_index])), "f"(dummy)
                        : /* no clobbers */);
            }

            // mask[i] = reg
            for (int i = thread_id_in_grp; i < PIM_RF_SIZE;
                    i += threads_per_pim_grp) {
                uint64_t mem_index = INDEX(chip, bank, col + i);

                asm volatile ("st.cs.global.u16 [%0], %1;"
                        : /* no outputs */
                        : "l"(&(mem_rows[mask_row].val[mem_index])), "h"(store)
                        : /* no clobbers */);
            }

            __threadfence();
        }
    }
}
